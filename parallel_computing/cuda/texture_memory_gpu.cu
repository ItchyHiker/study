#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 10
#define N 10

// first argument: data type of texture elements
// second argument: types of texture reference which can be one-dimensional, two-dimensional...
// third argument: read mode, optional 
texture <float, 1, hipReadModeElementType> textureRef;
__global__ void gpu_texture_memory(int n, float *d_out)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < N) {
        float temp = tex1D(textureRef, float(idx));
        d_out[idx] = temp;
    }
}

int main()
{
    int num_blocks = N / NUM_THREADS + ((N % NUM_THREADS) ? 1 : 0);
    float *d_out;

    hipMalloc((void**)&d_out, sizeof(float)*N);
    float h_out[N], h_in[N];
    for (int i = 0; i < N; i++) {
        h_in[i] = float(i);
    }
    
    // Define cuda array which is dedicated to textures compared to normal array
    hipArray *cu_array;
    hipMallocArray(&cu_array, &textureRef.channelDesc, N, 1);
    // copy data to cuda array
    // 0, 0 meaning starting from the top left corner
    hipMemcpyToArray(cu_array, 0, 0, h_in, N*sizeof(float), hipMemcpyHostToDevice);

    // bind a texture to the CUDA array
    hipBindTextureToArray(textureRef, cu_array);
    
    gpu_texture_memory << <num_blocks, NUM_THREADS>> > (N, d_out);

    hipMemcpy(h_out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);
    printf("Use of texture memory on GPU: \n");
    for (int i = 0; i < N; i++) {
        printf("Texture element at %d is: %f\n", i, h_out[i]);
    }

    hipFree(d_out);
    hipFreeArray(cu_array);
    hipUnbindTexture(textureRef);

}
