#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 500

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c)
{
    // Getting block index of current kernel
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    
    while (tid < N)
    {
        d_c[tid] = d_a[tid] + d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main()
{
    int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, N*sizeof(int));
    hipMalloc((void**)&d_b, N*sizeof(int));
    hipMalloc((void**)&d_c, N*sizeof(int));
    for (int i = 0; i < N; i++) {
        h_a[i] = 2 * i * i;
        h_b[i] = i;
    }
    hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice);
    
    gpuAdd << <512, 512>> > (d_a, d_b, d_c);
    
    hipMemcpy(h_c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    int correct = 1;
    for (int i = 0; i < N; i++) {
        if ((h_a[i] + h_b[i]) != h_c[i]) {
            // printf("h_a[%d] = %d, h_b[%d] = %d, h_c[%d] = %d\n", 
            //        i, h_a[i], i, h_b[i], i, h_c[i]);
            correct = 0;
        }
    }

    if (correct == 1) 
        printf("GPU has computed sum correctedly\n");
    else
        printf("GPU has failed to compute sum\n");
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
