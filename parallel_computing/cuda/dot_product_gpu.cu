#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 1024
#define threadsPerBlock 512

__global__ void gpu_dot(float *d_a, float *d_b, float *d_c)
{
    __shared__ float partial_sum[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int index = threadIdx.x;
    float sum = 0;
    while (tid < N) {
        sum += d_a[tid]*d_b[tid];
        tid += blockDim.x*gridDim.x;
    }

    partial_sum[index] = sum;

    __syncthreads();
    
    int i = blockDim.x / 2;
    while (i != 0) {
        if (index < i)
            partial_sum[index] += partial_sum[index+i];
        __syncthreads(); // after modification to shared memory
        i /= 2;
    }

    if (index == 0)
        d_c[blockIdx.x] = partial_sum[0];
}

int main()
{
    float h_a[N], h_b[N], h_c, *h_partial_sum;
    float *d_a, *d_b, *d_partial_sum;
    int block_calc = (N + threadsPerBlock - 1) / threadsPerBlock;
    int blocks_per_grid = (32 < block_calc ? 32 : block_calc);
    h_partial_sum = (float *)malloc(blocks_per_grid * sizeof(float));

    // allocate the memory on the device
    hipMalloc((void**)&d_a, N*sizeof(float));
    hipMalloc((void**)&d_b, N*sizeof(float));
    hipMalloc((void**)&d_partial_sum, blocks_per_grid*sizeof(float));

    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = 2;
    }

    hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);

    gpu_dot << <blocks_per_grid, threadsPerBlock>> > (d_a, d_b, d_partial_sum);
    
    // copy array back to host memory
    hipMemcpy(h_partial_sum, d_partial_sum, blocks_per_grid*sizeof(float), 
            hipMemcpyDeviceToHost);
    h_c = 0;
    // each block have a separate answer to be stored in the global memory so 
    // that it is indexed by the block ID
    // calculate final dot product
    for (int i = 0; i < blocks_per_grid; i++) {
        h_c += h_partial_sum[i];
    }
    printf("The computed dot product is: %f \n", h_c);
#define cpu_sum(x) (x*(x+1))
    if (h_c == cpu_sum((float)(N-1)))
    {
        printf("The dot product computed by GPU is correct\n");
    }
    else
    {
        printf("Error in dot product computation");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_partial_sum);
}

