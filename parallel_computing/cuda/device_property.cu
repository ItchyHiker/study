#include <memory>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstring> // memset

int main(void)
{
    int device;
    hipDeviceProp_t device_property;
    hipGetDevice(&device);
    printf("ID of device: %d\n", device);
    memset(&device_property, 0, sizeof(hipDeviceProp_t));
    device_property.major = 1;
    device_property.minor = 3;
    hipChooseDevice(&device, &device_property);
    printf("ID of device which supports double precision is: %d\n", device);
    hipSetDevice(device);
}
