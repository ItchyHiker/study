#include <memory>
#include <iostream>
#include <hip/hip_runtime.h>

int main(void)
{
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    std::cout << "There are " << device_count << " gpus on this computer" << std::endl;
}
