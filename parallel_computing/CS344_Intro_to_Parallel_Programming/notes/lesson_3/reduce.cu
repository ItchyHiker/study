#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void globalMem_reduce_kernel(float *d_out, float *d_in)
{
    int ttid = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) { // if threadIdx.x is on the left half
            d_in[ttid] += d_in[ttid + s];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        d_out[blockIdx.x] = d_in[ttid];
    }
}


__global__ void sharedMem_reduce_kernel(float *d_out, float *d_in)
{
    // shared data is allocated in the kernel call: 3rd argument
    extern __shared__ float shared_data[];
    int ttid = threadIdx.x + blockDim.x * threadIdx.x;
    int tid = threadIdx.x;

    // load shared memory from global memory
    shared_data[tid] = d_in[ttid];
    __syncthreads();

    // reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid+s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = shared_data[0];
    }
}

void reduce(float *d_out, float *d_intermediate, float *d_in, int size, 
        bool useSharedMem)
{
    // assumption 1: size is not greater than maxThreadsPerBlock**2
    // assumption 2: size is a multiple of maxThreadsPerBlock

    const int maxThreadsPerBlock = 1024;
    int threads = maxThreadsPerBlock;
    // int blocks = size % maxThreadsPerBlock ? 
    //    (size / maxThreadsPerBlock + 1) : (size / maxThreadsPerBlock);
    int blocks = size / maxThreadsPerBlock;

    if (useSharedMem) {
        sharedMem_reduce_kernel<<<blocks, threads, threads*sizeof(float)>>>(d_intermediate, d_in);
    } else {
        globalMem_reduce_kernel<<<blocks, threads>>>(d_intermediate, d_in);
    }

    threads = blocks;
    blocks = 1;

    if (useSharedMem) {
        sharedMem_reduce_kernel<<<blocks, threads, threads*sizeof(float)>>>(d_out, d_intermediate);
    } else {
        globalMem_reduce_kernel<<<blocks, threads>>>(d_out, d_intermediate);
    }
}


int main(int argc, char **argv)
{
    // --- checking whether there is a device --- //
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cerr << "No GPUs found" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    // --- get properties of device --- //
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProps;
    if (hipGetDeviceProperties(&deviceProps, dev) == 0) {
        std::cout << "Using device:" << dev << "\n";
        std::cout << deviceProps.name << "\n";
        std::cout << "Global memory: " << deviceProps.totalGlobalMem << "\n";
        std::cout << "Compute v:" << static_cast<int>(deviceProps.major) << "."
            << static_cast<int>(deviceProps.minor) << std::endl;
        std::cout << "Clock:" << static_cast<int>(deviceProps.clockRate) << std::endl;
    }

    const int ARRAY_SIZE = 1 << 20;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE];
    float sum = 0.0f;
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = -1.0f + static_cast<float>(random()) / (static_cast<float>(RAND_MAX)/2.0f);
        sum += h_in[i];
    }

    float *d_in, *d_intermediate, *d_out;
    hipMalloc((void **)&d_in, ARRAY_BYTES);
    hipMalloc((void **)&d_intermediate, ARRAY_BYTES);
    hipMalloc((void **)&d_out, sizeof(float));

    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    int whichKernel = 0;
    if (argc == 2) {
        whichKernel = atoi(argv[1]);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    switch (whichKernel) {
        case 0:
            std::cout << "Global memory reduce" << "\n";
            hipEventRecord(start, 0);
            for (int i = 0; i < 100; i++) {
                reduce(d_out, d_intermediate, d_in, ARRAY_SIZE, false);
            }
            hipEventRecord(stop, 0);
            break;
        case 1:
            std::cout << "Shared memory reduce" << "\n";
            hipEventRecord(start, 0);
            for (int i = 0; i < 100; i++) {
                reduce(d_out, d_intermediate, d_in, ARRAY_SIZE, true);
            }
            hipEventRecord(stop, 0);
            break;
        default:
            std::cerr << "No kernel run!" << std::endl;
            exit(EXIT_FAILURE);
    }

    // calculate elapsedTime
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    elapsed /= 100.0f;

    float h_out;
    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);
    
    std::cout << "Everage time elapsed:" << elapsed << std::endl;
    std::cout << "Host result:" << sum << ", device result:" << h_out << std::endl;

    hipFree(d_in);
    hipFree(d_intermediate);
    hipFree(d_out);
}
